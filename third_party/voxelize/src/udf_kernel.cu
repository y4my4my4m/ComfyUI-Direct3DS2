#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


struct Point3D {
    float x, y, z;
};

struct Triangle {
    Point3D v0, v1, v2;
};
__device__ Point3D cross(const Point3D& v1, const Point3D& v2) {
    Point3D result;
    result.x = v1.y * v2.z - v1.z * v2.y;
    result.y = v1.z * v2.x - v1.x * v2.z;
    result.z = v1.x * v2.y - v1.y * v2.x;
    return result;
}

// Compute the dot product of two vectors
__device__ float dot(const Point3D& v1, const Point3D& v2) {
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

// Subtract two 3D points (vector subtraction)
__device__ Point3D subtract(const Point3D& p1, const Point3D& p2) {
    Point3D result = {p1.x - p2.x, p1.y - p2.y, p1.z - p2.z};
    return result;
}
__device__ float magnitude(const Point3D &v) {
	    return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}
__device__ bool is_identical(const Point3D & p1,  const Point3D & p2){
	Point3D check = subtract(p1, p2);
	if(check.x==0 && check.y == 0 && check.z == 0)
		return true;
	return false;
}

// Compute the squared distance between two points
__device__ float squaredDistance(const Point3D& p1, const Point3D& p2) {
    return (p1.x - p2.x) * (p1.x - p2.x) +
           (p1.y - p2.y) * (p1.y - p2.y) +
           (p1.z - p2.z) * (p1.z - p2.z);
}
__device__ Point3D normalize(Point3D v){
	float len = sqrtf(dot(v, v));
	if (len ==0)
		return v;
	float scale = 1 / len;
	Point3D result = {v.x * scale, v.y * scale, v.z * scale};
       return result;	
}

__device__ float point_to_line_distance(const Point3D &p, const Point3D &v0, const Point3D &v1) {
    // Direction vector of the line
    Point3D d = subtract(v1, v0);

    // Vector from v0 to point p
    Point3D v0_to_p = subtract(p, v0);

    // Scalar projection of v0_to_p onto the direction vector d
    float t = dot(v0_to_p, d) / dot(d, d);

    Point3D closest_point;

    // Check where the projection falls
    if (t < 0) {
        // Projection falls before v0, so the closest point is v0
        closest_point = v0;
    } else if (t > 1) {
        // Projection falls beyond v1, so the closest point is v1
        closest_point = v1;
    } else {
        // Projection falls within the segment, compute the projection point
        closest_point.x = v0.x + t * d.x;
        closest_point.y = v0.y + t * d.y;
        closest_point.z = v0.z + t * d.z;
    }

    // Calculate the distance between p and the closest point
    Point3D closest_to_p = subtract(p, closest_point);
    return magnitude(closest_to_p);
}

// Compute the distance between a point and a triangle face
__device__ float pointToTriangleDistance(const Point3D& queryPoint, const Point3D& v0, const Point3D& v1, const Point3D& v2, bool inverse=false) {
    // Edge vectors
    Point3D edge0 = subtract(v1, v0);
    Point3D edge1 = subtract(v2, v0);
    if (is_identical(v0, v1) && is_identical(v0, v2))
	    return sqrtf(squaredDistance(queryPoint, v0));
    if (is_identical(v0, v1))
	    return point_to_line_distance(queryPoint, v0, v2);
    if (is_identical(v0, v2))
	    return point_to_line_distance(queryPoint, v0, v1);
    if (is_identical(v1, v2))
	    return point_to_line_distance(queryPoint, v0, v1);
    // Normal vector to the triangle plane
    Point3D normal = cross(edge0, edge1);
    if (inverse)
        normal = cross(edge1, edge0);
    
    // Vector from v0 to queryPoint
    Point3D queryVec = subtract(queryPoint, v0);
    if (dot(normal, normal)==0)
	    return sqrtf(dot(queryVec, queryVec));
    normal = normalize(normal);
    //return 1.0;
    
    // Project the query point onto the triangle's plane
    float distanceToPlane = dot(normal, queryVec); // / sqrtf(dot(normal, normal));
    
// return fabsf(distanceToPlane);
    Point3D projectionPoint = {
        queryPoint.x - distanceToPlane * normal.x,
        queryPoint.y - distanceToPlane * normal.y,
        queryPoint.z - distanceToPlane * normal.z
    };
    // Check if the projection point is inside the triangle using barycentric coordinates
    edge0 = subtract(v0, v1);
    edge1 = subtract(v1, v2);
    Point3D edge2 = subtract(v2, v0);
    Point3D projVec0 = subtract(v0, projectionPoint);
    Point3D projVec1 = subtract(v1, projectionPoint);
    Point3D projVec2 = subtract(v2, projectionPoint);
    Point3D c0 = cross(edge0, projVec0);
    Point3D c1 = cross(edge1, projVec1);
    Point3D c2 = cross(edge2, projVec2);
    if (dot(c0, c1) > 0 && dot(c1, c2) > 0 && dot(c0, c2) > 0)
        return fabsf(distanceToPlane);

    // Otherwise, return the minimum distance to the triangle's edges
    float minEdgeDistance = 1e6f;
    minEdgeDistance = fmin(minEdgeDistance, point_to_line_distance(queryPoint, v0, v1));
    minEdgeDistance = fmin(minEdgeDistance, point_to_line_distance(queryPoint, v0, v2));
    minEdgeDistance = fmin(minEdgeDistance, point_to_line_distance(queryPoint, v1, v2));
    
    
    return minEdgeDistance;
}


__device__ void updateUDF(Triangle t, int* udf, const int DIM, const float threshold) {
    // Compute the bounding box of the triangle
    float minX = fminf(fminf(t.v0.x, t.v1.x), t.v2.x);
    float minY = fminf(fminf(t.v0.y, t.v1.y), t.v2.y);
    float minZ = fminf(fminf(t.v0.z, t.v1.z), t.v2.z);
    float maxX = fmaxf(fmaxf(t.v0.x, t.v1.x), t.v2.x);
    float maxY = fmaxf(fmaxf(t.v0.y, t.v1.y), t.v2.y);
    float maxZ = fmaxf(fmaxf(t.v0.z, t.v1.z), t.v2.z);

    // Convert bounding box to grid coordinates
    int iMin = max(0, (int)floorf((minX + 0.5)  * (DIM-1)));
    int jMin = max(0, (int)floorf((minY + 0.5)  * (DIM-1)));
    int kMin = max(0, (int)floorf((minZ + 0.5)  * (DIM-1)));
    int iMax = min(DIM - 1, (int)floorf((maxX + 0.5)  * (DIM-1)));
    int jMax = min(DIM - 1, (int)floorf((maxY + 0.5)  * (DIM-1)));
    int kMax = min(DIM - 1, (int)floorf((maxZ + 0.5)  * (DIM-1)));

    int range = (int)(threshold + 1);
    
    // Make the bounding box larger than the original
    iMax = min(DIM - 1, iMax + range);
    iMin = max(0, iMin - range);
    jMax = min(DIM - 1, jMax + range);
    jMin = max(0, jMin - range);
    kMax = min(DIM - 1, kMax + range);
    kMin = max(0, kMin - range);

    // Update the valid grids within the bounding box
    for (int i = iMin; i <= iMax; ++i) {
        for (int j = jMin; j <= jMax; ++j) {
            for (int k = kMin; k <= kMax; ++k) {
                int idx = i * DIM * DIM + j * DIM + k;
        
        // Compute the distance from the query point to the triangle
                Point3D queryPoint = {(float)i/(DIM-1) - 0.5, (float)j/(DIM-1) - 0.5, (float)k/(DIM-1) -0.5};
                float distance = pointToTriangleDistance(queryPoint, t.v0, t.v1, t.v2);
                float distance2 = pointToTriangleDistance(queryPoint, t.v0, t.v1, t.v2, true);
	        if (distance < threshold / DIM or  distance2 < threshold / DIM){
		//distance = distance2;
		     int int_dist = (int)(distance * 10000000);
                     atomicMin(&udf[idx], int_dist);
		}
	    }
    
        }
    }
}

__global__ void compute_udf_kernel(float* vertices, int* faces, int * udf, int numTriangles, const int DIM, const float threshold) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < numTriangles) {
        int f0 = faces[t * 3 + 0];
        int f1 = faces[t * 3 + 1];
        int f2 = faces[t * 3 + 2];
        Point3D v0 = {vertices[f0 * 3 + 0], vertices[f0 * 3 + 1], vertices[f0 * 3 + 2]};
        Point3D v1 = {vertices[f1 * 3 + 0], vertices[f1 * 3 + 1], vertices[f1 * 3 + 2]};
        Point3D v2 = {vertices[f2 * 3 + 0], vertices[f2 * 3 + 1], vertices[f2 * 3 + 2]};
        Triangle triangle = {v0, v1, v2};
        updateUDF(triangle, udf, DIM, threshold);
    }
}

void compute_valid_udf_cuda(float* vertices, int* faces, int* udf, int numTriangles, const int DIM=512, const float threshold=8) {
    int blockSize = 256;
    int gridSize = (numTriangles + blockSize - 1) / blockSize;

    // Launch the kernel
    compute_udf_kernel<<<gridSize, blockSize>>>(vertices, faces, udf, numTriangles, DIM, threshold);
}

